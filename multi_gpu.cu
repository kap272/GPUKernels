#include <stdio.h>
#include <hip/hip_runtime.h>

float* generate_random_matrix(int rows, int cols) {
    float* M = (float*) malloc(sizeof(float) * rows * cols); 
    for (int i = 0; i < rows * cols; i++) {
        M[i] = rand() % 10;
    }
    return M;

}

float* allocate_matrix(int rows, int cols) {
    float* vec = (float*) malloc(sizeof(float) * rows * cols); 
    return vec;
}

bool matrices_are_equal(float* A, float *B, int A_rows, int A_cols, int B_rows, int B_cols) {
    if (A_rows != B_rows || A_cols != B_cols) {
        return false;
    }

    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_cols; j++) {
            if (A[i * A_cols + j] != B[i * B_cols + j]) {
                return false;
            }
        }
    }

    return true;
}

void print_matrix(float* M, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", M[cols * i + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void mat_mul(float* A, float* B, float* C, int A_rows, int A_cols, int B_cols) {
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < B_cols; j++) {
            float C_val = 0.0;
            for (int l = 0; l < A_cols; l++) {
                C_val += A[i * A_cols + l] * B[l * B_cols + j];
            }
            C[i * B_cols + j] = C_val;
        }
    }
}

__global__ void mat_mul_knl_naive(float* A, float* B, float* C, int A_rows, int A_cols, int B_cols) {
    int row = blockDim.y * blockIdx.y + threadIdx.y; 
    int col = blockDim.x * blockIdx.x + threadIdx.x; 

    if (row < A_rows && col < B_cols) {
        float C_val = 0.0;
        for (int inner_idx = 0; inner_idx < A_cols; inner_idx++) {
            C_val += A[row * A_cols + inner_idx] * B[inner_idx * B_cols + col];
        }
        C[row * B_cols + col] += C_val;

    }
}

int main(int argc, char** argv) {
    int gpu_count;
    bool print_results = false;
    hipError_t err = hipGetDeviceCount(&gpu_count);
    if (err != hipSuccess) {
        printf("failed to count gpus; "); printf(hipGetErrorString(err));
    }
    printf("found %d GPUs\n", gpu_count);
    printf("hello world\n");

    for (int i = 0; i < gpu_count; i++){
        for (int j = 0; j < gpu_count; j++){
            if (i != j) {
                int can_connect = 0;
                hipDeviceCanAccessPeer(&can_connect, i, j);
                if (can_connect){
                    printf("can connect gpus %d and %d as peers \n", i, j);
                } else {
                    printf("cannot connect gpus %d and %d as peers \n", i, j);
                }
            }
        }
    }


    int A_rows = atoi(argv[1]);
    int A_cols = atoi(argv[2]);
    int B_cols = atoi(argv[3]);

    int A_size = sizeof(float) * A_rows * A_cols;
    int B_size = sizeof(float) * A_cols * B_cols;
    int C_size = sizeof(float) * A_rows * B_cols;

    float* A = generate_random_matrix(A_rows, A_cols); 
    float* B = generate_random_matrix(A_cols, B_cols); 

    float* C_ref = allocate_matrix(A_rows, B_cols);

    // establish ground truth
//    print_matrix(A, A_rows, A_cols);
//    print_matrix(B, A_cols, B_cols);
    mat_mul(A, B, C_ref, A_rows, A_cols, B_cols);
    //print_matrix(C_ref, A_rows, B_cols);


    // run kernel f
    int N0 = A_cols/2;
    int N1 = A_cols - N0;

    float *A0_d=nullptr, *B0_d=nullptr, *C0_d=nullptr;
    float *A1_d=nullptr, *B1_d=nullptr, *C1_d=nullptr;


    hipSetDevice(0);
    hipMalloc(&A0_d, sizeof(float)* A_rows * A_cols);
    hipMalloc(&B0_d, sizeof(float)* B_cols * N0);
    hipMalloc(&C0_d, sizeof(float) * A_rows * N0);
    hipMemcpy(A0_d, A, sizeof(float) * A_rows * B_cols, hipMemcpyHostToDevice);
    hipMemcpy(B0_d, B, sizeof(float) * B_cols * N0, hipMemcpyHostToDevice);

    hipSetDevice(1);
    hipMalloc(&A1_d, sizeof(float)* A_rows * A_cols);
    hipMalloc(&B1_d, sizeof(float)* B_cols * N0);
    hipMalloc(&C1_d, sizeof(float)* A_rows * N0);
    hipMemcpy(A1_d, A, sizeof(float)*A_rows*A_cols, hipMemcpyHostToDevice);
    hipMemcpy(B1_d, B + (size_t) B_cols * N0, sizeof(float)*B_cols*N1, hipMemcpyHostToDevice);


    float* C_h_f = allocate_matrix(A_rows, B_cols);

    float* A_d;
    float* B_d;
    float* C_d_f = allocate_matrix(A_rows, B_cols);

    hipMalloc(&A_d, A_size);
    hipMalloc(&B_d, B_size); 
    hipMalloc(&C_d_f, C_size); 

    hipMemcpy(A_d, A, A_size, hipMemcpyHostToDevice); 
    hipMemcpy(B_d, B, B_size, hipMemcpyHostToDevice); 

    float f_time;
    hipEvent_t f_start, f_stop;    
    hipEventCreate(&f_start);
    hipEventCreate(&f_stop);
    hipEventRecord(f_start, 0);

    int block_width = 16;
    dim3 f_grid_dims(1 + A_cols/block_width, 1 + A_rows/block_width);
    dim3 f_block_dims(block_width, block_width);

    mat_mul_knl_naive<<<f_grid_dims, f_block_dims>>>(A_d, B_d, C_d_f, A_rows, A_cols, B_cols);
    
    hipEventRecord(f_stop, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&f_time, f_start, f_stop);

    hipMemcpy(C_h_f, C_d_f, C_size, hipMemcpyDeviceToHost); 
    hipFree(C_d_f);

    if (!matrices_are_equal(C_ref, C_h_f, A_rows, B_cols, A_rows, B_cols)) {
        printf("kernel f is wrong!\n");
    } else {
        printf("kernel f is not wrong \n");
    }
    if (print_results) {
        printf("f result: \n");
        print_matrix(C_h_f, A_rows, B_cols);
    }

    free(C_h_f);
}
